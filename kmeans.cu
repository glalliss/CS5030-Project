#include <hip/hip_runtime.h>

#include <ctime>
#include <fstream>
#include <iostream>
#include <limits>
#include <regex>
#include <sstream>
#include <string>
#include <vector>
#include <chrono>
const int k = 5;
const int epochs = 100;
struct Point {
    double x, y, z; // coordinates
    int cluster;    // no default cluster
    double minDist; // default infinite distance to nearest cluster
    // Initialize a point
    Point() :
        x(0.0), y(0.0), z(0.0), cluster(-1), minDist(std::numeric_limits<double>::max()) {}
    Point(double x, double y, double z) :
        x(x), y(y), z(z), cluster(-1), minDist(std::numeric_limits<double>::max()) {}
    
};
// Computes the (square) euclidean distance between this point and another
__device__ double distance(Point p1, Point p2) {
    return (p1.x - p2.x) * (p1.x - p2.x) +
           (p1.y - p2.y) * (p1.y - p2.y) +
           (p1.z - p2.z) * (p1.z - p2.z);
}

// kernel functiin to assign clusters to points
__global__ void assignPointsToClusters(Point* points, int numPoints, Point* centroids, int k, double maxout) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numPoints) {
        double minDist = maxout;
        int clusterId = -1;

        for (int i = 0; i < k; ++i) {
            double dist = distance(points[index], centroids[i]);
            if (dist < minDist) {
                minDist = dist;
                clusterId = i;
            }
        }

        points[index].minDist = minDist;
        points[index].cluster = clusterId;
    }
}

// helper kernel function to compute the mean for each cluster
__global__ void computeNewCentroids(Point* points, int numPoints, Point* centroids, int k, int* nPoints, double* sumX, double* sumY, double* sumZ) {
    int clusterId = blockIdx.x * blockDim.x + threadIdx.x;
    if (clusterId < k) {
        for (int index = 0; index < numPoints ; index++)
        {
            if(points[index].cluster == clusterId){
                nPoints[clusterId] += 1;
                sumX[clusterId] += points[index].x;
                sumY[clusterId] += points[index].y;
                sumZ[clusterId] += points[index].z;

            }
        }
    }
}
// kernel function to compute the mean for each cluster and update its centroid
__global__ void updateNewCentroids(Point* centroids,int* nPoints, double* sumX, double* sumY, double* sumZ) {
    int clusterId = blockIdx.x * blockDim.x + threadIdx.x;
    centroids[clusterId].x = sumX[clusterId] / nPoints[clusterId];
    centroids[clusterId].y = sumY[clusterId] / nPoints[clusterId];
    centroids[clusterId].z = sumZ[clusterId] / nPoints[clusterId];
}


// Reads in the data.csv file into a vector of points and return vector of points
std::vector<Point> readcsv()
{
    std::vector<Point> points;
    std::ifstream file("tracks_features.csv");
    std::string line;
    int danceabilityIndex = 9;
    int energyIndex = 10;
    int valenceIndex = 18;
    while (getline(file, line))
    {
        std::stringstream lineStream(line);
        std::vector<std::string> columns;
        while (!lineStream.eof())
        {
            std::string column;
            getline(lineStream, column, ',');
            columns.push_back(column);
            // Handle cases where a column contains a comma inside double quotes or a list
            if (columns.back().front() == '"' && std::count(columns.back().begin(), columns.back().end(), '"') % 2 != 0)
            {
                while (columns.back().front() == '"' && std::count(columns.back().begin(), columns.back().end(), '"') % 2 != 0)
                {
                    std::string nextColumn;
                    getline(lineStream, nextColumn, ',');
                    columns.back() += "," + nextColumn;
                    // std::cout << columns.back() << std::endl;
                }
            }
        }
        try
        {
            // std::cout << columns[danceabilityIndex] << "    " << columns[energyIndex] << "    " << columns[valenceIndex] << std::endl;
            double x, y, z;
            // Convert specific columns into a double to create a Point and add it to the points vector
            x = stod(columns[danceabilityIndex]);
            y = stod(columns[energyIndex]);
            z = stod(columns[valenceIndex]);
            points.push_back(Point(x, y, z));
        }
        catch (const std::invalid_argument& e)
        {
            // std::cerr << "Invalid argument: " << e.what() << std::endl;
            std::cerr << "Skipping first line with column names." << std::endl;
        }

        //std::cout << "done reading!" << std::endl;
    }
    // The points vector should/will have ~1.2M points to be used with the kMeansClustering function
    std::cout << "done reading!" << std::endl;
    return points;
}



int main() {
    std::vector<Point> data = readcsv();
    std::vector<Point>* points = &data;

    int n = points->size();
    int numPoints = n;
    std::cout << n << std::endl;
    // Randomly initialise centroids
    // The index of the centroid within the centroids vector represents the cluster label.
    std::vector<Point> centroids_data;
    srand(100);
    for (int i = 0; i < k; ++i)
    {
        centroids_data.push_back(points->at(rand() % n));
    }
    std::vector<Point>* centroids = &centroids_data;

    // Allocate GPU memory
    Point* d_points;
    hipMalloc((void**)&d_points, sizeof(Point) * numPoints);
    hipMemcpy(d_points, points->data(), sizeof(Point) * n, hipMemcpyHostToDevice);

    Point* d_centroids;
    hipMalloc((void**)&d_centroids, sizeof(Point) * k);
    hipMemcpy(d_centroids, centroids->data(), sizeof(Point) * k, hipMemcpyHostToDevice);

    int* d_nPoints;
    hipMalloc((void**)&d_nPoints, sizeof(int) * k);
    hipMemset(d_nPoints, 0, sizeof(int) * k);

    double* d_sumX;
    hipMalloc((void**)&d_sumX, sizeof(double) * k);
    hipMemset(d_sumX, 0, sizeof(double) * k);

    double* d_sumY;
    hipMalloc((void**)&d_sumY, sizeof(double) * k);
    hipMemset(d_sumY, 0, sizeof(double) * k);

    double* d_sumZ;
    hipMalloc((void**)&d_sumZ, sizeof(double) * k);
    hipMemset(d_sumZ, 0, sizeof(double) * k);

    std::cout << "Running algorithm for " << epochs << " epochs" << std::endl;
    auto start_time = std::chrono::high_resolution_clock::now();

    // Run k-means clustering on GPU
    for (int i = 0; i < epochs; ++i) {

        hipDeviceSynchronize();
        // Assign points to clusters
        assignPointsToClusters<<<ceil(1204025.0 / 256), 256>>>(d_points, numPoints, d_centroids, k, std::numeric_limits<double>::max());

        // Synchronize to ensure the previous kernel is finished
        hipDeviceSynchronize();

        // Compute new centroids
        hipMemset(d_nPoints, 0, sizeof(int) * k);
        hipMemset(d_sumX, 0, sizeof(double) * k);
        hipMemset(d_sumY, 0, sizeof(double) * k);
        hipMemset(d_sumZ, 0, sizeof(double) * k);

        computeNewCentroids<<<1,k>>>(d_points, numPoints, d_centroids, k, d_nPoints, d_sumX, d_sumY, d_sumZ);

        // Synchronize to ensure the previous kernel is finished
        hipDeviceSynchronize();

        updateNewCentroids<<<k, 1>>>(d_centroids,d_nPoints, d_sumX, d_sumY, d_sumZ);

        hipDeviceSynchronize();

    }
    // Update the data points with their centroids on the host
    hipMemcpy(points->data(), d_points, sizeof(Point) * numPoints, hipMemcpyDeviceToHost);
    auto end_time = std::chrono::high_resolution_clock::now();
    auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
    long time = elapsed_time.count();
    double secs = ((double)time) / (1000 * 1000);
    std::cout << "Duration : " << secs << std::endl;
    // Write to csv
    std::cout << "Writing to CSV" << std::endl;
    std::ofstream myfile;
    myfile.open("output_shared_gpu.csv");
    myfile << "x,y,z,c" << std::endl;
    for (std::vector<Point>::iterator it = points->begin(); it != points->end(); ++it)
    {
        myfile << it->x << "," << it->y << "," << it->z << "," << it->cluster << std::endl;
    }
    myfile.close();

    // Free GPU memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_nPoints);
    hipFree(d_sumX);
    hipFree(d_sumY);
    hipFree(d_sumZ);
    std::cout << "Finished successfully" << std::endl;

    return 0;
}
